#include "hip/hip_runtime.h"

#ifdef __HIPCC__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include "cudaUtils.hpp"
#include "defines.hpp"

namespace MarchingCubes {


    __global__ void 
    __launch_bounds__(512)
    computeDensitiesKernel(float x0, float y0, float z0,
            unsigned int W, unsigned int H, unsigned int L, 
            float h,
            hipSurfaceObject_t densitiesSurface, 
            float t) {
        
        unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
        unsigned int iz = blockIdx.z * blockDim.z + threadIdx.z;

        if(ix >= W || iy >= H || iz >= L)
            return;

        /*surf3Dwrite(__float2half_rn(0.5f), densitiesSurface, ix/2*sizeof(float), iy, iz, hipBoundaryModeTrap);*/
        /*if(ix < W/2)*/
            /*surf3Dwrite(1.0f, densitiesSurface, ix*sizeof(float), iy, iz, hipBoundaryModeTrap);*/
        /*else*/
            /*surf3Dwrite(0.0f, densitiesSurface, ix*sizeof(float), iy, iz, hipBoundaryModeTrap);*/
        /*float dx = float(ix)/W;*/
        /*float dy = float(iy)/H;*/
        /*float dz = float(iz)/L;*/
        surf3Dwrite(sin(ix/2*h)*sin(iy/4*h)*sin(iz*h/8) , densitiesSurface, ix*sizeof(float), iy, iz, hipBoundaryModeTrap);
    }

    __host__ void callComputeDensitiesKernel(
        float x0, float y0, float z0,
        unsigned int W, unsigned int H, unsigned int L, 
        float h, 
        hipSurfaceObject_t densitiesSurface) {
       
        dim3 blockDim(8,8,8);
        dim3 gridDim(
                (W+blockDim.x-1)/blockDim.x, 
                (H+blockDim.y-1)/blockDim.y, 
                (L+blockDim.z-1)/blockDim.z);

        static float t = 0.0f;
        static bool pos = true;
        if(pos) {
            t += 0.01;
            if(t > 1.0f) {
                pos = false;
                t = 1.0f;
            }
        }
        else {
            t -= 0.01;
            if(t < 0.0f) {
                t = 0.0f;
                pos = true;
            }
        }

        printf("frame !\n");

        computeDensitiesKernel<<<gridDim,blockDim,0>>>(x0,y0,z0,W,H,L,h,densitiesSurface,t);
        CHECK_KERNEL_EXECUTION();
    }
}

#endif /* ifdef __HIPCC__ */

