#include "hip/hip_runtime.h"

#include "implicitFunc.h"
#include <cstdio>
#include <cassert>
#include "cudaUtils.hpp"

__device__ __host__ float d1(float x, float y, float z) { return x; }
__device__ __host__ float d2(float x, float y, float z) { return y; }
__device__ __host__ float d3(float x, float y, float z) { return z; }

__device__ __host__ float op1(float d1, float d2) { return d1+d2; } 
__device__ __host__ float op2(float d1, float d2) { return d1-d2; } 

const unsigned int nDensityFunctions = 3u;
const unsigned int nOperatorFunctions = 2u;

const densityFunction density_functions_p_h[nDensityFunctions] = {d1,d2,d3};
const operatorFunction operator_functions_p_h[nOperatorFunctions] = {op1,op2};

__device__ densityFunction density_functions_p_d[nDensityFunctions];
__device__ operatorFunction operator_functions_p_d[nOperatorFunctions];

__global__ void initPointersKernel() {
    density_functions_p_d[0] = d1;
    density_functions_p_d[1] = d2;
    density_functions_p_d[2] = d3;
    operator_functions_p_d[0] = op1;
    operator_functions_p_d[1] = op2;
}

std::map<operatorFunction, operatorFunction> operatorFunctionPointers;
std::map<densityFunction, densityFunction> densityFunctionPointers;

__host__ void initPointers() {

    //densities
    densityFunction density_functions_p_d_h[nDensityFunctions] = {d1,d2,d3};

    printf("Init pointers kernel :\n");
    initPointersKernel<<<1,1>>>();
    checkKernelExecution();

    CHECK_CUDA_ERRORS(hipMemcpyFromSymbol(
                &density_functions_p_d_h, 
                HIP_SYMBOL(density_functions_p_d), 
                nDensityFunctions*sizeof(densityFunction),
                0,hipMemcpyDeviceToHost));
   
    printf("Densities :\n");
    for (unsigned int i = 0u; i < nDensityFunctions; i++) {
        densityFunctionPointers.insert(
                std::pair<densityFunction,densityFunction>(density_functions_p_h[i], density_functions_p_d_h[i])
                );        
        printf("\tDensity %i: %p \t %p\n", i, density_functions_p_h[i], density_functions_p_d_h[i]);
    }
   
    //operators
    operatorFunction operator_functions_p_d_h[nOperatorFunctions];
    CHECK_CUDA_ERRORS(hipMemcpyFromSymbol(
                operator_functions_p_d_h, 
                HIP_SYMBOL(operator_functions_p_d), 
                nOperatorFunctions*sizeof(operatorFunction),
                0,hipMemcpyDeviceToHost));
    
    printf("Operators :\n");
    for (unsigned int i = 0u; i < nOperatorFunctions; i++) {
        operatorFunctionPointers.insert(
                std::pair<operatorFunction,operatorFunction>(operator_functions_p_h[i], operator_functions_p_d_h[i])
                );        
        printf("\tOperator %i: %p \t %p\n", i, operator_functions_p_h[i], operator_functions_p_d_h[i]);
    }
}
