#include "hip/hip_runtime.h"

#include "spinlock.hpp"


__host__ __device__ hd_atomic_flag::hd_atomic_flag() {
#ifdef __CUDA_ARCH__
    gpu = 0;
#else
    cpu.clear();
#endif
}

__host__ __device__ hd_atomic_flag::hd_atomic_flag(const hd_atomic_flag& af) {
#ifdef __CUDA_ARCH__
    gpu = af.gpu;
#else
    cpu = af.cpu;
#endif
}

__host__ __device__ hd_atomic_flag& hd_atomic_flag::operator= (const hd_atomic_flag& af) {
#ifdef __CUDA_ARCH__
    gpu = af.gpu;
#else
    cpu = af.cpu;
#endif
    return *this;
}

__host__ __device__ hd_atomic_flag::~hd_atomic_flag() {
#ifdef __CUDA_ARCH__
    gpu = 0;
#else
    cpu.clear();
#endif
}

__host__ __device__ Spinlock::Spinlock() : _flag() {
}
__host__ __device__ Spinlock::Spinlock(const Spinlock& sl) :
    _flag(sl._flag) {

    }

__host__ __device__ Spinlock& Spinlock::operator= (const Spinlock& sl) {
    this->_flag = sl._flag;
    return *this;
}

__host__ __device__ Spinlock::~Spinlock() {
}

__host__ __device__ void Spinlock::lock() {
#ifdef __CUDA_ARCH__
    while(atomicCAS(&_flag.gpu, 0, 1) != 0);
#else
    while(_flag.cpu.test_and_set(std::memory_order_acquire));
#endif
}

__host__ __device__ bool Spinlock::try_lock() {
#ifdef __CUDA_ARCH__
    return atomicCAS(&_flag.gpu, 0, 1) == 0;
#else
    return _flag.cpu.test_and_set(std::memory_order_acquire);
#endif
}

__host__ __device__ void Spinlock::unlock() {
#ifdef __CUDA_ARCH__
    atomicExch(&_flag.gpu, 1);
#else
    _flag.cpu.clear(std::memory_order_release); 
#endif
}

